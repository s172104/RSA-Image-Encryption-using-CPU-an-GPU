#include "hip/hip_runtime.h"
// Adam Zi�kowski: Image encryption using RSA alogrithm with GPU approach.
// STB library used at MIT License, license added to project repository.
//
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"


uint64_t Modulus; 
uint64_t PrivateExponent;
uint64_t PublicExponent;

// Kernel uses iterative modular exponentiation algorithm, each one takes single pixel to encrypt or decrypt it
__global__ void ModExpKernel(uint64_t* original, uint64_t* product, uint64_t exponent, uint64_t mod, int* size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < *size){
		uint64_t result = 1;      // Initialize result 
		uint64_t x = original[i];
		
	    x = x % mod;  // Checking if x is more than or equal to mod
	  
	    while (exponent > 0) 
	    { 
	        // I multiply x with result if exp is odd, 
	        if (exponent & 1) 
	            result = (result * x) % mod; 
	  
	        // exp must be even now  
	        exponent = exponent >> 1; // exp = exp/2 
	        x = (x*x) % mod;   
	    } 
	    product[i] = result; 
	}
}


// Test if chosen number is prime   
bool ifPrimary(uint64_t n) {
    bool flag = false;
    if (n != 2 && n % 2 == 0) {
        flag = true;
    }
    else {
        for (uint64_t d = 3; d * d <= n; d++) {
            if (n % d == 0) {
                flag = true;
                break;
            }
        }
    }
    return flag;
}


// Algorithm for finding greatest common divisor used for finding main part of private key
uint64_t ExtendedEuclideanAlgorithm(uint64_t a, uint64_t b, uint64_t* x, uint64_t* y) {

    if (a == 0) {
        *x = 0;
        *y = 1;
        return b;
    }
    uint64_t x1, y1;
    uint64_t gcd = ExtendedEuclideanAlgorithm(b % a, a, &x1, &y1);
    *x = y1 - (b / a) * x1;
    *y = x1;
    return gcd;

}


// Function searching for greatest common divisor
uint64_t GCD(uint64_t x, uint64_t y) {

    uint64_t gcd = NAN;

    for (uint64_t i = 1; i <= x && i <= y; i++) {      // Keep executing loop as long as i is less or equal to one of factors
        if (x % i == 0 && y % i == 0)                  // GCD divides both x and y
            gcd = i;
    }

    return gcd;
}


// Generating two key pairs using RSA key generation algorithm
static void GenerateKeyPair(void) {

    uint64_t p, q, n, d, Phi, gcd, y;
    uint16_t upper_limit = 65535; //18446744073709551615;
    uint16_t lower_limit = 55000;  //8446744073709551615;


    // 1. Generate a pair of large, random primes p and q
    do {
        p = lower_limit + (rand() % (upper_limit - lower_limit));
    } while (ifPrimary(p));
    do {
        q = lower_limit + (rand() % (upper_limit - lower_limit));
    } while (ifPrimary(q) || p == q);

    // 2. Compute the modulus n = pq
    n = p * q;

    // 3. Calculate Phi using Euler's totient function
    Phi = (p - 1) * (q - 1);

    // 4. Find e that is relatively prime to Phi
    uint64_t e = 3;

    while (e < Phi) {
        if (GCD(e, Phi) == 1)
            break;
        else
            e += 2;
    }

    // 5. Compute the private exponent d from e, p and q.
    gcd = ExtendedEuclideanAlgorithm(e, Phi, &d, &y);

    // 6. Output(e, n) as the public key and (d, n) as the private key

    if ((d * e) % Phi == 1) {  // Test to ensure legitimacy of a key komponents
        printf("Git\n");
        Modulus = n;
        PublicExponent = e;
        PrivateExponent = d;
    }
    else {
        GenerateKeyPair();
    }
}


int main()
{
    srand(time(NULL));
    int width, height, bpp;
    uint64_t* encrypted64, * decrypted64;
    uint8_t* encrypted, * decrypted, * original;
    uint8_t* rgb_image = stbi_load("Lenna.png", &width, &height, &bpp, CHANNEL_NUM);
    
    uint64_t* rgb_image64, buff[4];
    int size = (width * height * CHANNEL_NUM);
    int size64 = size / 2;
    rgb_image64 = new uint64_t[size64];

	// Merging 2 8bit pixels into 64bit structure
    for (int i = 0; i < size64; i++) {
        for (int k = 0; k < 2; k++) {
            buff[k] = 0b0;
            buff[k] = rgb_image[i * 2 + k];
            buff[k] = buff[k] << (8 - k * 8);
        }
        rgb_image64[i] = buff[0] | buff[1];

    }

    GenerateKeyPair();

    printf("Klucz publiczny: %I64u, %I64u\nKlucz prywatny: %I64u, %I64u", PublicExponent, Modulus, PrivateExponent, Modulus);
    printf("\nRozpoczynam szyfrowanie\n");

     // Memory allocation for 8 bit structures for holding final results
    encrypted = new uint8_t[size];
    decrypted = new uint8_t[size];
    original = new uint8_t[size];

    // Allocation of 64 bit structures for holding computing results and said computing
    encrypted64 = new uint64_t[size64];
    decrypted64 = new uint64_t[size64];

	// Here I create device variables
	uint64_t* dev_encrypted64, * dev_decrypted64,* dev_rgb_image64;
	int* dev_size64;
	
	// Device memory allocation with data copy
	hipMalloc((void**)&dev_rgb_image64, size64*sizeof(uint64_t));
	hipMalloc((void**)&dev_encrypted64, size64*sizeof(uint64_t));
	hipMalloc((void**)&dev_decrypted64, size64*sizeof(uint64_t));
	hipMalloc((void**)&dev_size64, sizeof(int));
	hipMemcpy(dev_rgb_image64, rgb_image64, size64 * sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_size64, &size64, sizeof(int), hipMemcpyHostToDevice);
	
	// Kernel launched for encryption and decryption, both have their own result matrix for result storage 
	ModExpKernel<<<1024,1024>>>(dev_rgb_image64, dev_encrypted64, PublicExponent, Modulus, dev_size64);
	ModExpKernel<<<1024,1024>>>(dev_encrypted64, dev_decrypted64, PrivateExponent, Modulus, dev_size64);
	
	// Copying all images to host
	hipMemcpy(encrypted64, dev_encrypted64, size64 * sizeof(uint64_t), hipMemcpyDeviceToHost);
	hipMemcpy(decrypted64, dev_decrypted64, size64 * sizeof(uint64_t), hipMemcpyDeviceToHost);
	hipMemcpy(rgb_image64, dev_rgb_image64, size64 * sizeof(uint64_t), hipMemcpyDeviceToHost);
	
 
    // Sampling merged pixels to 8 bit form with binary mask and bit shift
    for (int i = 0; i < size64; i++) {
        for (int k = 0; k < 2; k++) {
            buff[k] = 255;
            buff[k] = buff[k] << (8 - k * 8);
            decrypted[i * 2 + k] = (uint8_t)((decrypted64[i] & buff[k]) >> (8 - k * 8));
            encrypted[i * 2 + k] = (uint8_t)((encrypted64[i] & buff[k]) >> (8 - k * 8));
            original[i * 2 + k] = (uint8_t)((rgb_image64[i] & buff[k]) >> (8 - k * 8));
        }
    }

    // Writing images to .png files
    stbi_write_png("original.png", width, height, CHANNEL_NUM, original, width * CHANNEL_NUM);
    stbi_write_png("decrypted.png", width, height, CHANNEL_NUM, decrypted, width * CHANNEL_NUM);
    stbi_write_png("encrypted.png", width, height, CHANNEL_NUM, encrypted, width * CHANNEL_NUM);

    // Releasing memory
    delete(rgb_image);
    delete(rgb_image64);
    delete(encrypted);
    delete(encrypted64);
    delete(decrypted);
    delete(decrypted64);
    delete(original);

    return 0;
}


